/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iterator>
#include <iomanip>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
using namespace std;


/**
 * CiiUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorQuantumComputing(float *qbit_input_quantum_state, float *qbit_quantum_gate, float *qbit_output_quantum_state,int qbit_value,  int numElements)
{
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

        if(i < numElements && i % int (__powf(2,(qbit_value+1)))<int (__powf(2,(qbit_value))))
        {
            qbit_output_quantum_state[i] = (qbit_quantum_gate[0] * qbit_input_quantum_state[i] + qbit_quantum_gate[1] * qbit_input_quantum_state[i | (1 << qbit_value)]);
            qbit_output_quantum_state[i | (1 << qbit_value)] = (qbit_quantum_gate[2] * qbit_input_quantum_state[i] + qbit_quantum_gate[3] * qbit_input_quantum_state[i | (1 << qbit_value)]);

        }
    
}

/**
 * Host main routine
 */
int
main(void)
{


    FILE * FP;
    FP=fopen("input.txt","r");
    int number_of_lines;
    char element;
    float qbit_quantum_gate_temp[2][2];


    if(FP==NULL){
        cout<<"File not found"<<endl;
        return 0;
    }

    while (EOF != (element=getc(FP))) {
        if ('\n' == element)
            number_of_lines=number_of_lines+1;
    }

    float* qbit_input_quantum_state = new float [(number_of_lines-3)];
    float* qbit_output_quantum_state = new float [(number_of_lines-4)];

    FP=fopen("input.txt","r");

    int i=0;
    while(fscanf(FP, "%f %f", &qbit_quantum_gate_temp[i][0], &qbit_quantum_gate_temp[i][1]) != EOF)
    {
        i++;
        if (i>1)
        {
            i = 0;
            while (fscanf(FP, "%f ", &qbit_input_quantum_state[i]) != EOF)
            {
                i++;
            }
            break;
        }
    }



    int qbit_input_quantum_state_length=number_of_lines-4;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = number_of_lines-4;
    size_t size = numElements  * sizeof(float);
    size_t size_gate = 4 * sizeof(float);
    size_t size_out = numElements * sizeof(float);



int qbit_value=qbit_input_quantum_state[numElements];
//cout<<"last element is  "<<qbit_input_quantum_state[numElements]<<endl;
//cout<<"Number of lines "<<number_of_lines<<endl;
   
 // Allocate the host input vector A

    // Allocate the host input vector B
    float* qbit_quantum_gate=new float [4];
    qbit_quantum_gate[0]=qbit_quantum_gate_temp[0][0];
    qbit_quantum_gate[1]=qbit_quantum_gate_temp[0][1];
    qbit_quantum_gate[2]=qbit_quantum_gate_temp[1][0];
    qbit_quantum_gate[3]=qbit_quantum_gate_temp[1][1];
    // Allocate the host output vector C
    // Verify that allocations succeeded
    if (qbit_input_quantum_state == NULL || qbit_quantum_gate == NULL || qbit_output_quantum_state == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    // Initialize the host input vectors
    // Allocate the device input vector A
    float *d_qbit_input_quantum_state = NULL;
    err = hipMalloc((void **)&d_qbit_input_quantum_state, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_qbit_input_quantum_state  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device input vector B
    float *d_qbit_quantum_gate = NULL;
    err = hipMalloc((void **)&d_qbit_quantum_gate, size_gate);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_qbit_quantum_gate  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_qbit_output_quantum_state = NULL;
    err = hipMalloc((void **)&d_qbit_output_quantum_state, size_out);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_qbit_output_quantum_state (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    err = hipMemcpy(d_qbit_input_quantum_state, qbit_input_quantum_state, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy qbit_input_quantum_state from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_qbit_quantum_gate, qbit_quantum_gate, size_gate, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy qbit_quantum_gate from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorQuantumComputing<<<blocksPerGrid, threadsPerBlock>>>(d_qbit_input_quantum_state, d_qbit_quantum_gate, d_qbit_output_quantum_state,qbit_value, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    err = hipMemcpy(qbit_output_quantum_state, d_qbit_output_quantum_state, size_out, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy qbit_output_quantum_state from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

 //   for(i=0;i<4;i++)
   // {
     //   cout<<fixed<<setprecision(3)<<qbit_quantum_gate[i]<<endl;
//}


//cout<<"---------------------------------------------------"<<endl;
  //  for(i=0;i<=qbit_input_quantum_state_length;i++)
   // {
     //   cout<<fixed<<setprecision(3)<<qbit_input_quantum_state[i]<<endl;
//}
//cout<<"---------------------------------------------------"<<endl;
    for(i=0;i<qbit_input_quantum_state_length;i++)
    {
        cout<<fixed<<setprecision(3)<<qbit_output_quantum_state[i]<<endl;
    }
    // Verify that the result vector is correct
    // Free device global memory
    err = hipFree(d_qbit_input_quantum_state);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device d_qbit_input_quantum_state  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_qbit_quantum_gate);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device d_qbit_quantum_gate (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_qbit_output_quantum_state);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device d_qbit_output_quantum_state  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free host memory
    free(qbit_input_quantum_state);
    free(qbit_quantum_gate);
    free(qbit_output_quantum_state);


    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

